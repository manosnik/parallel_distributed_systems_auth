/*-----------------------------------------------------------
The program has been run at Google Colab with the following orders
-------------------------------------------------------------
!apt-get --purge remove cuda nvidia* libnvidia-*
!dpkg -l | grep cuda- | awk '{print $2}' | xargs -n1 dpkg --purge
!apt-get remove cuda-*
!apt autoremove
!apt-get update
--------------------------------------------------------------
!wget https://developer.nvidia.com/compute/cuda/9.2/Prod/local_installers/cuda-repo-ubuntu1604-9-2-local_9.2.88-1_amd64 -O cuda-repo-ubuntu1604-9-2-local_9.2.88-1_amd64.deb
!dpkg -i cuda-repo-ubuntu1604-9-2-local_9.2.88-1_amd64.deb
!apt-key add /var/cuda-repo-9-2-local/7fa2af80.pub
!apt-get update
!apt-get install cuda-9.2
--------------------------------------------------------------
!nvcc --version
--------------------------------------------------------------
%load_ext nvcc_plugin
--------------------------------------------------------------
and at the AUTH's cluster with the known procedure*/

//%%cu necessary in Google Colab


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BILLION 1E9

#define N 2000 //N=M
#define M 2000
#define K 31     //must be K%2=1
#define b 20     //max b around 10 because of the size of shared memory
#define BV1 4000 //BV1 should not be more than 1000x smaller than N*M
#define BV2 2000
#define BV3 2000 //1.The number of threads of every block in version 3 multiplied with b should not exceed the size of shared memory and \
                   //2.The number of columns(=threadsPerBlock*b) that everuy block computes should follow .. M%Block Col==0

//function of serial code

int sign(int a, int t, int c, int d, int e)
{

    int temp = 0;
    temp = a + t + c + d + e;
    if (temp > 0)
    {
        return 1;
    }
    else if (temp < 0)
    {
        return -1;
    }
    else if (temp == 0)
    {
        printf("Error in sign function\n");
        return 0;
    }
    return 7;
}

//function of v1

__global__ void isingModel_v1(int *initialArray, int *secondaryArray)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    int i = index / M;
    int j = index % N;

    secondaryArray[i * N + j] = initialArray[((i - 1 + N) % N) * N + (j % M)] + initialArray[(i % N) * N + ((j - 1 + M) % M)] + initialArray[(i % N) * N + (j % M)] + initialArray[((i + 1) % N) * N + (j % M)] + initialArray[(i % N) * N + ((j + 1) % M)];

    if (secondaryArray[i * N + j] > 0)
    {
        secondaryArray[i * N + j] = 1;
    }
    else if (secondaryArray[i * N + j] < 0)
    {
        secondaryArray[i * N + j] = -1;
    }
    else if (secondaryArray[i * N + j] == 0)
    {
        secondaryArray[i * N + j] = 3;
    }

    __syncthreads();
}

//function v2

__global__ void isingModel_v2(int *initialArray, int *secondaryArray)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int rowStart = (int)(index / (int)(M / b)) * b;
    int colStart = (int)(index % (int)(M / b)) * b;

    for (int i = rowStart; i < rowStart + b & i < N; i++)
    {
        for (int j = colStart; j < colStart + b & j < M; j++)
        {
            secondaryArray[i * N + j] = initialArray[((i - 1 + N) % N) * N + (j % M)] + initialArray[(i % N) * N + ((j - 1 + M) % M)] + initialArray[(i % N) * N + (j % M)] + initialArray[((i + 1) % N) * N + (j % M)] + initialArray[(i % N) * N + ((j + 1) % M)];

            if (secondaryArray[i * N + j] > 0)
            {
                secondaryArray[i * N + j] = 1;
            }
            else if (secondaryArray[i * N + j] < 0)
            {
                secondaryArray[i * N + j] = -1;
            }
            else if (secondaryArray[i * N + j] == 0)
            {
                secondaryArray[i * N + j] = 3;
            }
        }
    }
    __syncthreads();
}

//function of v3

__global__ void isingModel_v3(int *initialArray, int *secondaryArray)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int rowStart = (int)(index / (int)(N / b)) * b;
    int colStart = (int)(index % (int)(M / b)) * b;

    int threadsPerBlock = ((N * M) / (b * b)) / BV3;
    int cudaArrJ = threadsPerBlock * b;
    int cudaArrI = M / (threadsPerBlock * b);

    __shared__ int cudaSharedArr[M * N / BV3];

    for (int i = rowStart; i < rowStart + b & i < N; i++)
    {
        for (int j = colStart; j < colStart + b & j < M; j++)
        {

            cudaSharedArr[(i % b) * cudaArrJ + j % cudaArrJ] = initialArray[i * M + j];
        }
    }

    __syncthreads();

    int a, c, d, e;
    for (int i = rowStart; i < rowStart + b & i < N; i++)
    {
        for (int j = colStart; j < colStart + b & i < M; j++)
        {

            if (i - 1 == rowStart - 1)
            {
                a = initialArray[((i - 1 + N) % N) * M + (j % M)];
            }
            else
            {
                a = cudaSharedArr[(((i - 1 + N) % N) % b) * cudaArrJ + j % cudaArrJ];
            }

            if (i + 1 == rowStart + b)
            {
                e = initialArray[((i + 1) % N) * M + (j % M)];
            }
            else
            {
                e = cudaSharedArr[(((i + 1) % N) % b) * cudaArrJ + j % cudaArrJ];
            }

            if (threadIdx.x == 0 && j == colStart)
            {
                c = initialArray[(i % N) * M + ((j - 1 + M) % M)];
            }
            else
            {
                c = cudaSharedArr[(i % b) * cudaArrJ + ((j - 1 + M) % M) % cudaArrJ];
            }

            if (threadIdx.x == blockDim.x - 1 && j == colStart + b - 1)
            {

                d = initialArray[(i % N) * M + ((j + 1) % M)];
            }
            else
            {
                d = cudaSharedArr[(i % b) * cudaArrJ + ((j + 1) % M) % cudaArrJ];
            }

            secondaryArray[i * M + j] = a + c + cudaSharedArr[(i % b) * cudaArrJ + j % cudaArrJ] + e + d;

            if (secondaryArray[i * M + j] > 0)
            {
                secondaryArray[i * M + j] = 1;
            }
            else if (secondaryArray[i * M + j] < 0)
            {
                secondaryArray[i * M + j] = -1;
            }
            else if (secondaryArray[i * M + j] == 0)
            {
                secondaryArray[i * M + j] = 3;
            }
        }
    }
}

int main(int argc, char *argv[])
{

    const int size = N * M * sizeof(int);

    // declare the initial and the secondry 2D array
    int **initialArray_v0 = (int **)malloc(N * sizeof(int *));
    int **secondaryArray_v0 = (int **)malloc(N * sizeof(int *));

    for (int i = 0; i < N; i++)
    {
        initialArray_v0[i] = (int *)malloc(M * sizeof(int));
        secondaryArray_v0[i] = (int *)malloc(M * sizeof(int));
    }

    // initialize the fist array for all versions
    srand(time(0));
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            initialArray_v0[i][j] = rand() % 2;
            if (initialArray_v0[i][j] == 0)
            {
                initialArray_v0[i][j] = -1;
            }
        }
    }

    int *initialArray_v1 = (int *)malloc(N * M * sizeof(int *));
    int *initialArray_v2 = (int *)malloc(N * M * sizeof(int *));
    int *initialArray_v3 = (int *)malloc(N * M * sizeof(int *));

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            initialArray_v1[i * M + j] = initialArray_v0[i][j];
            initialArray_v2[i * M + j] = initialArray_v0[i][j];
            initialArray_v3[i * M + j] = initialArray_v0[i][j];
        }
    }
    //Checking that the 4 versions begin from the same starting point
    int fd = 0;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            if (initialArray_v0[i][j] != initialArray_v1[i * N + j] || initialArray_v0[i][j] != initialArray_v2[i * N + j] || initialArray_v0[i][j] != initialArray_v3[i * N + j])
            {
                printf("Erorr\n");
                fd = 1;
                break;
            }
        }
    }
    if (fd == 0)
    {
        printf("4 identical matrices\n");
    }

    //Start the clock

    struct timespec requestStart, requestEnd;
    clock_gettime(CLOCK_REALTIME, &requestStart);

    // Ising model of serial
    for (int k = 0; k < K; k++)
    {

        for (int i = 0; i < N; i++)
        {

            for (int j = 0; j < M; j++)
            {
                secondaryArray_v0[i][j] = sign(initialArray_v0[(i - 1 + N) % N][j], initialArray_v0[i][(j - 1 + M) % M], initialArray_v0[i][j], initialArray_v0[(i + 1) % N][j], initialArray_v0[i][(j + 1) % M]);
            }
        }

        // Copying the pointers
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < M; j++)
            {
                initialArray_v0[i][j] = secondaryArray_v0[i][j];
            }
        }
    }

    // Calculate time it took

    clock_gettime(CLOCK_REALTIME, &requestEnd);

    double accum = (requestEnd.tv_sec - requestStart.tv_sec) + (requestEnd.tv_nsec - requestStart.tv_nsec) / BILLION;
    printf("Time it took v0 :%lf seconds\n", accum);

    //version 1

    int threads_v1 = N * M / BV1;

    int *cudaInitialArray_v1, *cudaSecondaryArray_v1;

    hipMalloc((void **)&cudaInitialArray_v1, size);
    hipMalloc((void **)&cudaSecondaryArray_v1, size);

    clock_gettime(CLOCK_REALTIME, &requestStart);
    hipMemcpy(cudaInitialArray_v1, initialArray_v1, size, hipMemcpyHostToDevice);
    for (int k = 0; k < K; k++)
    {
        if (k % 2 == 0)
        {
            isingModel_v1<<<BV1, threads_v1>>>(cudaInitialArray_v1, cudaSecondaryArray_v1);
        }
        else
        {
            isingModel_v1<<<BV1, threads_v1>>>(cudaSecondaryArray_v1, cudaInitialArray_v1);
        }
    }
    hipMemcpy(initialArray_v1, cudaSecondaryArray_v1, size, hipMemcpyDeviceToHost);
    // Calculate time it took

    clock_gettime(CLOCK_REALTIME, &requestEnd);

    accum = (requestEnd.tv_sec - requestStart.tv_sec) + (requestEnd.tv_nsec - requestStart.tv_nsec) / BILLION;
    printf("Time it took v1 :%lf seconds\n", accum);

    hipFree(cudaInitialArray_v1);
    hipFree(cudaSecondaryArray_v1);

    //version 2

    int threads_v2 = ((N * M) / (b * b)) / BV2;

    int *cudaInitialArray_v2, *cudaSecondaryArray_v2;

    hipMalloc((void **)&cudaInitialArray_v2, size);
    hipMalloc((void **)&cudaSecondaryArray_v2, size);

    clock_gettime(CLOCK_REALTIME, &requestStart);

    hipMemcpy(cudaInitialArray_v2, initialArray_v2, size, hipMemcpyHostToDevice);

    for (int k = 0; k < K; k++)
    {
        if (k % 2 == 0)
        {
            isingModel_v2<<<BV2, threads_v2>>>(cudaInitialArray_v2, cudaSecondaryArray_v2);
        }
        else
        {
            isingModel_v2<<<BV2, threads_v2>>>(cudaSecondaryArray_v2, cudaInitialArray_v2);
        }
    }

    hipMemcpy(initialArray_v2, cudaSecondaryArray_v2, size, hipMemcpyDeviceToHost);

    // Calculate time it took

    clock_gettime(CLOCK_REALTIME, &requestEnd);
    accum = (requestEnd.tv_sec - requestStart.tv_sec) + (requestEnd.tv_nsec - requestStart.tv_nsec) / BILLION;
    printf("Time it took v2 :%lf seconds\n", accum);

    hipFree(cudaInitialArray_v2);
    hipFree(cudaSecondaryArray_v2);

    //version 3

    int threads_v3 = ((N * M) / (b * b)) / BV3;

    int *cudaInitialArray_v3, *cudaSecondaryArray_v3;

    hipMalloc((void **)&cudaInitialArray_v3, size);
    hipMalloc((void **)&cudaSecondaryArray_v3, size);

    clock_gettime(CLOCK_REALTIME, &requestStart);
    hipMemcpy(cudaInitialArray_v3, initialArray_v3, size, hipMemcpyHostToDevice);

    for (int k = 0; k < K; k++)
    {
        if (k % 2 == 0)
        {

            isingModel_v3<<<BV3, threads_v3>>>(cudaInitialArray_v3, cudaSecondaryArray_v3);
        }
        else
        {
            isingModel_v3<<<BV3, threads_v3>>>(cudaSecondaryArray_v3, cudaInitialArray_v3);
        }
    }
    hipMemcpy(initialArray_v3, cudaSecondaryArray_v3, size, hipMemcpyDeviceToHost);
    // Calculate time it took

    clock_gettime(CLOCK_REALTIME, &requestEnd);
    accum = (requestEnd.tv_sec - requestStart.tv_sec) + (requestEnd.tv_nsec - requestStart.tv_nsec) / BILLION;
    printf("Time it took v3 :%lf seconds\n", accum);

    hipFree(cudaInitialArray_v3);
    hipFree(cudaSecondaryArray_v3);

    int fb = 0, errorsC = 0;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            if (initialArray_v0[i][j] != initialArray_v1[i * M + j] || initialArray_v0[i][j] != initialArray_v2[i * M + j] || initialArray_v0[i][j] != initialArray_v3[i * M + j])
            {
                if (errorsC < 10)
                {
                    fb = 1;
                    if (initialArray_v0[i][j] != initialArray_v1[i * M + j])
                    {
                        printf("Erorr in v1\n");
                    }
                    if (initialArray_v0[i][j] != initialArray_v2[i * M + j])
                    {
                        printf("Erorr in v2\n");
                    }
                    if (initialArray_v0[i][j] != initialArray_v3[i * M + j])
                    {
                        printf("Erorr in v3\n");
                    }
                    errorsC++;
                }
                break;
            }
        }
    }
    if (fb == 0)
    {
        printf("4 identical matrices at end\n");
    }

    return 0;
}